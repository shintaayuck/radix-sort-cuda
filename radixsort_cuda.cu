#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

void rng(int* arr, int n) {
    int seed = 13516110; // Ganti dengan NIM anda sebagai seed.
    srand(seed);
    for(long i = 0; i < n; i++) {
        arr[i] = (int)rand();
    }
}

__global__
void generate_flags(int* arr, int n, int idx, int* flag) {

    int bit_test = 1 << idx;
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i+=stride) {
        if ((arr[i] & bit_test) == bit_test) {
            flag[i] = 0;
        } else {
            flag[i] = 1;
        }
    }
    return flag;
}

int* generate_index_down(int* flag, int n) {
    int* index_down = (int*) malloc(n * sizeof(int));
    index_down[0] = 0;

    for (int i = 1; i < n; i++) {
        index_down[i] = index_down[i-1] + flag[i-1];
    }

    return index_down;
}

int* generate_index_up(int* flag, int n) {
    int* index_up = (int*) malloc(n * sizeof(int));
    index_up[n-1] = n-1;
    for (int i = n-2; i >=0; i--) {
        int diff;
        if (flag[i+1]) {
            diff = 0;
        } else {
            diff = 1;
        }
        index_up[i] = index_up[i+1] - diff;
    }
    return index_up;
}

__global__
void permute(int* arr, int* temps, int* flag, int* index_down, int* index_up, int* arr_idx int n) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

    for (i = index; i < n; i+=stride) {
        if (flag[i]) {
            indexes[i] = index_down[i];
        } else {
            indexes[i] = index_up[i];
        }
    }
	hipDeviceSynchronize();

    for (i = index; i < n; i+=stride) {
        arr[indexes[i]] = temps[i];
    }

}

void split(int n, int idx, int* d_arr) {
    // assign flags
	int* flags, d_flags;
	hipMalloc(&d_flags, sizeof(int) * n);

	int block_size = 256; // harus bisa dibagi 32
	int num_blocks = block_size + n - 1;

    generate_flags<<<num_blocks, block_size>>>(d_arr, n, idx, d_flags);
	hipDeviceSynchronize();

	flags = (int*) malloc(sizeof(int)*n);
	hipMemcpy(flags, d_flags, sizeof(int), hipMemcpyDeviceToHost);

    int* index_down = generate_index_down(flag, n);
    int* index_up = generate_index_up(flag, n);

	int *d_temps, *d_arr_idx, *d_idx_down, *d_idx_up;
	hipMalloc(&d_temps, sizeof(int)*n);
	hipMemcpy(d_temps, d_arr, sizeof(int)*n, hipMemcpyDeviceToDevice);

	hipMalloc(&d_arr_idx, sizeof(int)*n);

	hipMalloc(&d_idx_down, sizeof(int)*n);
	hipMemcpy(d_idx_down, index_down, sizeof(int)*n, hipMemcpyHostToDevice);

	hipMalloc(&d_idx_up, sizeof(int)*n);
	hipMemcpy(d_idx_up, index_up, sizeof(int)*n, hipMemcpyHostToDevice);

    permute<<<num_blocks, block_size>>>(d_arr, d_temps, flag, d_index_down, d_index_up, d_arr_idx, n);
	hipDeviceSynchronize();

	hipFree(d_flags);
	hipFree(d_temps);
	hipFree(d_idx_down);
	hipFree(d_idx_up);
	hipFree(d_arr_idx);

	free(flags);
}


void radix_sort(int* arr, int n, int* d_arr) {

	hipMalloc(&d_arr, sizeof(int) * n);
	hipMemcpy(d_arr, sizeof(int) * n, hipMemcpyHostToDevice);

    for (int i=0; i<32; i++) {
      split(n, i, d_arr);
    }

	hipMemcpy(arr, d_arr, sizeof(int) * n, hipMemcpyDeviceToHost);
	hipFree(d_arr);
}

// A utility function to print an array
void print(int* arr, int n)
{
    for (int i = 0; i < n; i++)
        printf("%d ", arr[i]);
}


int main(int argc, char** argv) {
  // if (argc<2) {
  //   printf("Usage : radix_sort <array length>\n");
  //   return 0;
  // }


  // int n = atoi(argv[1]);
  int n = 1000;
  int* arr = (int*) malloc(sizeof(int)*n);

  rng(arr, n);
  clock_t start = clock();
  radix_sort(arr,n);
  print(arr, n);
  clock_t end = clock();
  double total_time = ((double) (end - start)) / (CLOCKS_PER_SEC / 1000);
  print(arr, n);
  printf("%f\n", total_time);


  // Tulis di file eksternal
  FILE *file = fopen("output.txt", "w");
  if (file == NULL)
  {
      printf("Error opening output.txt!\n");
      exit(1);
  }

  for (int i = 0; i < n; i++){
      fprintf(file, "%d\n", arr[i]);
  }

  fclose(file);
  free(arr);
}
